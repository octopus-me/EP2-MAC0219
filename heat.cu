#include "hip/hip_runtime.h"
  GNU nano 7.2                                                                                                                                                                                                                                                                                                   heat.cu                                                                                                                                                                                                                                                                                                            
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

#define WALL_TEMP 20.0
#define FIREPLACE_TEMP 100.0

#define FIREPLACE_START 3
#define FIREPLACE_END 7
#define ROOM_SIZE 10

// Kernel CUDA para calcular Jacobi na GPU
__global__ void jacobi_kernel(double *d_h, double *d_g, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        d_g[i * n + j] = 0.25 * (d_h[(i - 1) * n + j] + d_h[(i + 1) * n + j] +
                                 d_h[i * n + (j - 1)] + d_h[i * n + (j + 1)]);
    } else if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
        d_g[i * n + j] = d_h[i * n + j]; // Copiar diretamente os valores das bordas
    }
}

// Função para inicializar a matriz com condições de contorno
void initialize(double *h, int n) {
    int fireplace_start = (FIREPLACE_START * n) / ROOM_SIZE;
    int fireplace_end = (FIREPLACE_END * n) / ROOM_SIZE;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == 0 || i == n - 1 || j == 0 || j == n - 1) {
                h[i * n + j] = (i == n - 1 && j >= fireplace_start && j <= fireplace_end) ? FIREPLACE_TEMP : WALL_TEMP;
            } else {
                h[i * n + j] = 0.0;
            }
        }
    }
}

// Função Jacobi sequencial para execução na CPU
void jacobi_iteration_cpu(double *h, double *g, int n, int iter_limit) {
    for (int iter = 0; iter < iter_limit; iter++) {
        for (int i = 1; i < n - 1; i++) {
            for (int j = 1; j < n - 1; j++) {
                g[i * n + j] = 0.25 * (h[(i - 1) * n + j] + h[(i + 1) * n + j] +
                                       h[i * n + (j - 1)] + h[i * n + (j + 1)]);
            }
        }
        for (int i = 1; i < n - 1; i++) {
            for (int j = 1; j < n - 1; j++) {
                h[i * n + j] = g[i * n + j];
            }
        }
    }
}

// Função para calcular o tempo de execução na CPU
double calculate_elapsed_time(struct timespec start, struct timespec end) {
    double start_sec = (double)start.tv_sec * 1e9 + (double)start.tv_nsec;
    double end_sec = (double)end.tv_sec * 1e9 + (double)end.tv_nsec;
    return (end_sec - start_sec) / 1e9;
}

// Função para validar os resultados de CPU e GPU
void validate(double *h_cpu, double *h_gpu, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (fabs(h_cpu[i * n + j] - h_gpu[i * n + j]) > 1e-6) {
                printf("Discrepância encontrada em (%d, %d): CPU=%.6f, GPU=%.6f\n", i, j, h_cpu[i * n + j], h_gpu[i * n + j]);
                return;
            }
        }
    }
    printf("Validação bem-sucedida: CPU e GPU produzem os mesmos resultados.\n");
}

// Função principal
int main(int argc, char *argv[]) {
    if (argc < 5) {
        fprintf(stderr, "Uso: %s <n> <iter_limit> <threads_por_bloco> <blocos_por_grade>\n", argv[0]);
        return 1;
    }

    int n = atoi(argv[1]);
    int iter_limit = atoi(argv[2]);
    int threads_per_block = atoi(argv[3]);
    int blocks_per_grid = atoi(argv[4]);

    size_t size = n * n * sizeof(double);
    double *h_h = (double *)malloc(size);
    double *h_g = (double *)malloc(size);
    double *h_h_cpu = (double *)malloc(size);
    double *g_h_cpu = (double *)malloc(size);

    double *d_h, *d_g;
    hipMalloc(&d_h, size);
    hipMalloc(&d_g, size);

    initialize(h_h, n);
    memcpy(h_h_cpu, h_h, size); // Copiar os valores iniciais para a CPU

    hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);

    dim3 threads(threads_per_block, threads_per_block);
    dim3 blocks(blocks_per_grid, blocks_per_grid);

    // Executar Jacobi na CPU
    struct timespec start_cpu, end_cpu;
    clock_gettime(CLOCK_MONOTONIC, &start_cpu);
    jacobi_iteration_cpu(h_h_cpu, g_h_cpu, n, iter_limit);
    clock_gettime(CLOCK_MONOTONIC, &end_cpu);

    double elapsed_time_cpu = calculate_elapsed_time(start_cpu, end_cpu);
    printf("Tempo de execução na CPU: %.9f segundos\n", elapsed_time_cpu);

    // Executar Jacobi na GPU
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start_gpu);
    for (int iter = 0; iter < iter_limit; iter++) {
        jacobi_kernel<<<blocks, threads>>>(d_h, d_g, n);
        hipMemcpy(d_h, d_g, size, hipMemcpyDeviceToDevice);
    }
    hipEventRecord(stop_gpu);

    hipMemcpy(h_g, d_h, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop_gpu);

    float elapsed_time_gpu;
    hipEventElapsedTime(&elapsed_time_gpu, start_gpu, stop_gpu);
    printf("Tempo de execução na GPU: %.3f ms\n", elapsed_time_gpu);

    // Comparar os resultados
    validate(h_h_cpu, h_g, n);

    // Limpeza
    free(h_h);
    free(h_g);
    free(h_h_cpu);
    free(g_h_cpu);
    hipFree(d_h);
    hipFree(d_g);

    return 0;
}